#include "hip/hip_runtime.h"
// main.cu

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "kernel_2.cu"

int main() {
    // Seed for random number generation
    srand(42); // Fixed seed for reproducibility


    unsigned int N1;   // Input size (number of bits)
    unsigned int N2 = 20;  // Hidden layer size (adjusted for better learning)
    unsigned int N3 = 1;
    float learning_rate = 0.1f;
    unsigned int epochs = 10000;
    unsigned int threads_per_block = 256;

    // Open the output file
    FILE *output_file = fopen("gpu_cublas_output_1.txt", "a");
    if (output_file == NULL) {
        std::cerr << "Failed to open output file." << std::endl;
        return -1;
    }
    setvbuf(output_file, NULL, _IOLBF, 0);

    for(int i=11;i<26;i=i+1)
    {
        N1=i;

        // Hyperparameters
        unsigned long long total_samples = pow(2,N1);  // For 6 bits, 2^6 combinations
        unsigned long long train_size = total_samples*0.75;     // 75% for training
        // unsigned int test_size = total_samples - train_size;
       

        // Dimensions
        // Output size

        // Allocate host memory for data
        float *h_X = (float *)malloc(total_samples * N1 * sizeof(float));    // Input data
        float *h_Y = (float *)malloc(total_samples * sizeof(float));         // Labels

        // Generate dataset
        for (unsigned long long i = 0; i < total_samples; ++i) {
            unsigned long long num_ones = 0;
            for (unsigned long long j = 0; j < N1; ++j) {
                unsigned int bit = (i >> j) & 1;
                h_X[i * N1 + j] = (float)bit;
                num_ones += bit;
            }
            h_Y[i] = (num_ones % 2 == 1) ? 1.0f : 0.0f;
        }

        // Split data into training and test sets
        float *h_X_train = (float *)malloc(train_size * N1 * sizeof(float));
        float *h_Y_train = (float *)malloc(train_size * sizeof(float));
        // float h_X_test[test_size * N1];
        // float h_Y_test[test_size];
        if (h_X_train == NULL || h_Y_train == NULL) {
            std::cerr << "Failed to allocate host memory for training data." << std::endl;
            free(h_X); free(h_Y);
            return -1;
        }

        // Simple split
        memcpy(h_X_train, h_X, train_size * N1 * sizeof(float));
        memcpy(h_Y_train, h_Y, train_size * sizeof(float));
        // memcpy(h_X_test, h_X + train_size * N1, test_size * N1 * sizeof(float));
        // memcpy(h_Y_test, h_Y + train_size, test_size * sizeof(float));

        // Initialize weights and biases
        float *h_W1 = (float *)malloc(N1 * N2 * sizeof(float));          // Weights between input and hidden layer
        float *h_b1 = (float *)malloc(N2 * sizeof(float));               // Biases for hidden layer
        float *h_W2 = (float *)malloc(N2 * N3 * sizeof(float));          // Weights between hidden and output layer
        float *h_b2 = (float *)malloc(N3 * sizeof(float));                  // Biases for output layer

        if (h_W1 == NULL || h_b1 == NULL || h_W2 == NULL || h_b2 == NULL) {
            std::cerr << "Failed to allocate host memory for weights and biases." << std::endl;
            free(h_X); free(h_Y);
            free(h_X_train); free(h_Y_train);
            return -1;
        }

        // Xavier initialization
        float limit = sqrtf(6.0f / (N1 + N2));
        for (unsigned int i = 0; i < N1 * N2; ++i) {
            h_W1[i] = ((float)rand() / RAND_MAX) * 2 * limit - limit;
        }
        limit = sqrtf(6.0f / (N2 + N3));
        for (unsigned int i = 0; i < N2 * N3; ++i) {
            h_W2[i] = ((float)rand() / RAND_MAX) * 2 * limit - limit;
        }
        memset(h_b1, 0, N2 * sizeof(float));
        memset(h_b2, 0, N3 * sizeof(float));

        // Allocate device memory for training data
        float *d_X, *d_Y, *d_W1, *d_b1, *d_W2, *d_b2;
        float *d_Z1, *d_A1, *d_Z2, *d_A2;
        float *d_dW1, *d_db1, *d_dW2, *d_db2;
        float *d_dA1, *d_dZ1, *d_dZ2;

        hipMalloc((void**)&d_X, train_size * N1 * sizeof(float));
        hipMalloc((void**)&d_Y, train_size * sizeof(float));
        hipMalloc((void**)&d_W1, N1 * N2 * sizeof(float));
        hipMalloc((void**)&d_b1, N2 * sizeof(float));
        hipMalloc((void**)&d_W2, N2 * N3 * sizeof(float));
        hipMalloc((void**)&d_b2, N3 * sizeof(float));

        hipMalloc((void**)&d_Z1, train_size * N2 * sizeof(float));
        hipMalloc((void**)&d_A1, train_size * N2 * sizeof(float));
        hipMalloc((void**)&d_Z2, train_size * N3 * sizeof(float));
        hipMalloc((void**)&d_A2, train_size * N3 * sizeof(float));

        hipMalloc((void**)&d_dW1, N1 * N2 * sizeof(float));
        hipMalloc((void**)&d_db1, N2 * sizeof(float));
        hipMalloc((void**)&d_dW2, N2 * N3 * sizeof(float));
        hipMalloc((void**)&d_db2, N3 * sizeof(float));

        hipMalloc((void**)&d_dA1, train_size * N2 * sizeof(float));
        hipMalloc((void**)&d_dZ1, train_size * N2 * sizeof(float));
        hipMalloc((void**)&d_dZ2, train_size * N3 * sizeof(float));

        // Copy training data to device
        hipMemcpy(d_X, h_X_train, train_size * N1 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Y, h_Y_train, train_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W1, h_W1, N1 * N2 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b1, h_b1, N2 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W2, h_W2, N2 * N3 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b2, h_b2, N3 * sizeof(float), hipMemcpyHostToDevice);

        // Create cuBLAS handle
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        // Training loop
        for (unsigned int epoch = 0; epoch < epochs; ++epoch) {
            // Zero gradients
            hipMemset(d_dW1, 0, N1 * N2 * sizeof(float));
            hipMemset(d_db1, 0, N2 * sizeof(float));
            hipMemset(d_dW2, 0, N2 * N3 * sizeof(float));
            hipMemset(d_db2, 0, N3 * sizeof(float));

            // Forward pass
            forward_pass(handle, d_X, d_W1, d_b1, d_W2, d_b2, d_Z1, d_A1, d_Z2, d_A2, train_size, N1, N2, N3);

            // Backward pass
            backward_pass(handle, d_X, d_Y, d_W1, d_W2, d_b1, d_b2, d_Z1, d_A1, d_Z2, d_A2,
                        d_dW1, d_db1, d_dW2, d_db2, d_dA1, d_dZ1, d_dZ2,
                        train_size, N1, N2, N3);

            // Update weights and biases on the device
            unsigned int num_blocks_W1 = (N1 * N2 + threads_per_block - 1) / threads_per_block;
            unsigned int num_blocks_b1 = (N2 + threads_per_block - 1) / threads_per_block;
            unsigned int num_blocks_W2 = (N2 * N3 + threads_per_block - 1) / threads_per_block;
            unsigned int num_blocks_b2 = (N3 + threads_per_block - 1) / threads_per_block;

            update_parameters_kernel<<<num_blocks_W1, threads_per_block>>>(d_W1, d_dW1, learning_rate, N1 * N2, train_size);
            update_parameters_kernel<<<num_blocks_W2, threads_per_block>>>(d_W2, d_dW2, learning_rate, N2 * N3, train_size);
            update_parameters_kernel<<<num_blocks_b1, threads_per_block>>>(d_b1, d_db1, learning_rate, N2, train_size);
            update_parameters_kernel<<<num_blocks_b2, threads_per_block>>>(d_b2, d_db2, learning_rate, N3, train_size);

            // Optionally print loss and training accuracy every 1000 epochs
            // if ((epoch + 1) % 1000 == 0) {
            //     float h_loss = compute_loss(d_Y, d_A2, train_size);
            //     float training_accuracy = compute_accuracy(d_Y, d_A2, train_size);
            //     std::cout << "Epoch [" << epoch + 1 << "/" << epochs << "], Loss: " << h_loss
            //               << ", Training Accuracy: " << training_accuracy << "%" << std::endl;
            // }
    }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float trainingTime;
        hipEventElapsedTime(&trainingTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        printf("Training for %d epochs completed\n", epochs);
        // printf("******** GPU: Inputs = %d; Total Training time = %0.4f seconds ********\n",N1, trainingTime / 1000);
        fprintf(output_file, "******** GPU: Inputs = %d; Total Training time = %0.4f seconds ********\n",N1, trainingTime / 1000);
        fflush(output_file);

        // // Compute final training accuracy
        // float training_accuracy = compute_accuracy(d_Y, d_A2, train_size);
        // std::cout << "Final Training Accuracy: " << training_accuracy << "%" << std::endl;

        // // Testing the trained model
        // // Allocate device memory for test data
        // float *d_X_test, *d_Y_test, *d_Z1_test, *d_A1_test, *d_Z2_test, *d_A2_test;
        // hipMalloc((void**)&d_X_test, test_size * N1 * sizeof(float));
        // hipMalloc((void**)&d_Y_test, test_size * sizeof(float));
        // hipMalloc((void**)&d_Z1_test, test_size * N2 * sizeof(float));
        // hipMalloc((void**)&d_A1_test, test_size * N2 * sizeof(float));
        // hipMalloc((void**)&d_Z2_test, test_size * N3 * sizeof(float));
        // hipMalloc((void**)&d_A2_test, test_size * N3 * sizeof(float));

        // // Copy test data to device
        // hipMemcpy(d_X_test, h_X_test, test_size * N1 * sizeof(float), hipMemcpyHostToDevice);
        // hipMemcpy(d_Y_test, h_Y_test, test_size * sizeof(float), hipMemcpyHostToDevice);

        // // Forward pass on test data
        // forward_pass(handle, d_X_test, d_W1, d_b1, d_W2, d_b2, d_Z1_test, d_A1_test, d_Z2_test, d_A2_test,
        //              test_size, N1, N2, N3);

        // // Compute test accuracy
        // float test_accuracy = compute_accuracy(d_Y_test, d_A2_test, test_size);
        // std::cout << "Test Accuracy: " << test_accuracy << "%" << std::endl;

        // Clean up cuBLAS handle
        hipblasDestroy(handle);

        // Free device memory
        hipFree(d_X); hipFree(d_Y);
        hipFree(d_W1); hipFree(d_b1);
        hipFree(d_W2); hipFree(d_b2);
        hipFree(d_Z1); hipFree(d_A1);
        hipFree(d_Z2); hipFree(d_A2);
        hipFree(d_dW1); hipFree(d_db1);
        hipFree(d_dW2); hipFree(d_db2);
        hipFree(d_dA1); hipFree(d_dZ1); hipFree(d_dZ2);
    
        free(h_X); free(h_Y);
        free(h_X_train); free(h_Y_train);
        free(h_W1); free(h_b1);
        free(h_W2); free(h_b2);

        // hipFree(d_X_test); hipFree(d_Y_test);
        // hipFree(d_Z1_test); hipFree(d_A1_test);
        // hipFree(d_Z2_test); hipFree(d_A2_test);
    }

    fclose(output_file);

    return 0;
}
