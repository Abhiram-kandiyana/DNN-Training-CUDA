#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define NUM_EPOCHS 10000

// Xavier initialization
void xavier_init(int in_dim, int out_dim, float *W) {
    float limit = sqrtf(6.0f / (in_dim + out_dim));
    for (int i = 0; i < in_dim * out_dim; ++i) {
        W[i] = ((float)rand() / RAND_MAX) * 2 * limit - limit;
    }
}

// Generate parity data
void generate_parity_data(int N, int num_samples, float *X, float *Y) {
    for (int i = 0; i < num_samples; ++i) {
        int value = i;
        int parity = 0;
        for (int j = 0; j < N; ++j) {
            int bit = (value >> j) & 1;
            X[i * N + j] = (float)bit;
            parity ^= bit;
        }
        Y[i] = (float)parity;
    }
}

// CUDA kernel for weight updates
__global__ void update_weights(float *W, float *dW, float learning_rate, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        W[idx] -= learning_rate * dW[idx];
    }
}

// Helper function to create and set tensor descriptors
void create_and_set_tensor_descriptor(hipdnnTensorDescriptor_t *desc, int n, int c, int h, int w) {
    hipdnnCreateTensorDescriptor(desc);
    hipdnnSetTensor4dDescriptor(*desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w);
}

// Helper function to create and set filter descriptors
void create_and_set_filter_descriptor(hipdnnFilterDescriptor_t *desc, int k, int c, int h, int w) {
    hipdnnCreateFilterDescriptor(desc);
    hipdnnSetFilter4dDescriptor(*desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, h, w);
}

int main() {
    srand(time(NULL));

    for (int N = 2; N <= 25; ++N) {
        int input_size = N;
        int hidden_size = 20;
        int output_size = 1;
        int num_samples = 1 << N; // 2^N samples

        // Memory limitation check
        if (num_samples > (1 << 20)) { // Limit samples to avoid memory issues
            printf("Skipping N = %d due to memory constraints.\n", N);
            continue;
        }

        printf("Training for input size N = %d\n", N);

        // Allocate host memory
        float *h_X = (float *)malloc(num_samples * input_size * sizeof(float));
        float *h_Y = (float *)malloc(num_samples * sizeof(float));

        // Generate parity data
        generate_parity_data(N, num_samples, h_X, h_Y);

        // Allocate device memory
        float *d_X, *d_Y;
        hipMalloc((void **)&d_X, num_samples * input_size * sizeof(float));
        hipMalloc((void **)&d_Y, num_samples * sizeof(float));

        // Copy data to device
        hipMemcpy(d_X, h_X, num_samples * input_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Y, h_Y, num_samples * sizeof(float), hipMemcpyHostToDevice);

        // Initialize weights and biases
        float *h_W1 = (float *)malloc(input_size * hidden_size * sizeof(float));
        float *h_b1 = (float *)calloc(hidden_size, sizeof(float));
        float *h_W2 = (float *)malloc(hidden_size * output_size * sizeof(float));
        float *h_b2 = (float *)calloc(output_size, sizeof(float));

        xavier_init(input_size, hidden_size, h_W1);
        xavier_init(hidden_size, output_size, h_W2);

        // Allocate device memory for weights and biases
        float *d_W1, *d_b1, *d_W2, *d_b2;
        hipMalloc((void **)&d_W1, input_size * hidden_size * sizeof(float));
        hipMalloc((void **)&d_b1, hidden_size * sizeof(float));
        hipMalloc((void **)&d_W2, hidden_size * output_size * sizeof(float));
        hipMalloc((void **)&d_b2, output_size * sizeof(float));

        // Copy weights and biases to device
        hipMemcpy(d_W1, h_W1, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b1, h_b1, hidden_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W2, h_W2, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b2, h_b2, output_size * sizeof(float), hipMemcpyHostToDevice);

        // cuDNN handle
        hipdnnHandle_t cudnn;
        hipdnnCreate(&cudnn);

        // Tensor descriptors
        hipdnnTensorDescriptor_t input_desc, hidden_desc, output_desc;
        create_and_set_tensor_descriptor(&input_desc, num_samples, input_size, 1, 1);
        create_and_set_tensor_descriptor(&hidden_desc, num_samples, hidden_size, 1, 1);
        create_and_set_tensor_descriptor(&output_desc, num_samples, output_size, 1, 1);

        // Activation descriptor
        hipdnnActivationDescriptor_t activation_desc;
        hipdnnCreateActivationDescriptor(&activation_desc);
        hipdnnSetActivationDescriptor(activation_desc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0);

        // Convolution descriptor (reused for both layers)
        hipdnnConvolutionDescriptor_t conv_desc;
        hipdnnCreateConvolutionDescriptor(&conv_desc);
        hipdnnSetConvolution2dDescriptor(conv_desc,
                                        0, 0, // pad_h, pad_w
                                        1, 1, // stride_h, stride_w
                                        1, 1, // dilation_h, dilation_w
                                        HIPDNN_CROSS_CORRELATION,
                                        HIPDNN_DATA_FLOAT);

        // Filter descriptors
        hipdnnFilterDescriptor_t filter_desc1, filter_desc2;
        create_and_set_filter_descriptor(&filter_desc1, hidden_size, input_size, 1, 1);
        create_and_set_filter_descriptor(&filter_desc2, output_size, hidden_size, 1, 1);

        // Workspace for convolution
        size_t workspace_bytes_fwd1 = 0, workspace_bytes_fwd2 = 0;
        size_t workspace_bytes_bwd_filter1 = 0, workspace_bytes_bwd_filter2 = 0;
        size_t workspace_bytes_bwd_data1 = 0, workspace_bytes_bwd_data2 = 0;
        size_t workspace_bytes = 0;
        void *d_workspace = NULL;

        // Get workspace sizes
        hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                input_desc,
                                                filter_desc1,
                                                conv_desc,
                                                hidden_desc,
                                                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                                &workspace_bytes_fwd1);

        hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                hidden_desc,
                                                filter_desc2,
                                                conv_desc,
                                                output_desc,
                                                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                                &workspace_bytes_fwd2);

        hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn,
                                                       input_desc,
                                                       hidden_desc,
                                                       conv_desc,
                                                       filter_desc1,
                                                       HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                                       &workspace_bytes_bwd_filter1);

        hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn,
                                                       hidden_desc,
                                                       output_desc,
                                                       conv_desc,
                                                       filter_desc2,
                                                       HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                                       &workspace_bytes_bwd_filter2);

        hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn,
                                                     filter_desc2,
                                                     output_desc,
                                                     conv_desc,
                                                     hidden_desc,
                                                     HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                                     &workspace_bytes_bwd_data2);

        // Find the maximum required workspace size
        workspace_bytes = max(max(workspace_bytes_fwd1, workspace_bytes_fwd2),
                              max(max(workspace_bytes_bwd_filter1, workspace_bytes_bwd_filter2),
                                  workspace_bytes_bwd_data2));

        if (workspace_bytes > 0) {
            hipMalloc(&d_workspace, workspace_bytes);
        }

        // Allocate device memory for activations
        float *d_hidden, *d_output;
        hipMalloc((void **)&d_hidden, num_samples * hidden_size * sizeof(float));
        hipMalloc((void **)&d_output, num_samples * output_size * sizeof(float));

        // Allocate device memory for gradients
        float *d_delta_output, *d_delta_hidden;
        hipMalloc((void **)&d_delta_output, num_samples * output_size * sizeof(float));
        hipMalloc((void **)&d_delta_hidden, num_samples * hidden_size * sizeof(float));

        float *d_dW1, *d_db1, *d_dW2, *d_db2;
        hipMalloc((void **)&d_dW1, input_size * hidden_size * sizeof(float));
        hipMalloc((void **)&d_db1, hidden_size * sizeof(float));
        hipMalloc((void **)&d_dW2, hidden_size * output_size * sizeof(float));
        hipMalloc((void **)&d_db2, output_size * sizeof(float));

        // Create bias descriptors
        hipdnnTensorDescriptor_t bias_desc1, bias_desc2;
        create_and_set_tensor_descriptor(&bias_desc1, 1, hidden_size, 1, 1);
        create_and_set_tensor_descriptor(&bias_desc2, 1, output_size, 1, 1);

        // OpTensor descriptor for computing delta_output
        hipdnnOpTensorDescriptor_t op_desc;
        hipdnnCreateOpTensorDescriptor(&op_desc);
        hipdnnSetOpTensorDescriptor(op_desc, CUDNN_OP_TENSOR_SUB, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);

        // Start timing
        clock_t start = clock();

        // Training loop
        float alpha = 1.0f;
        float beta = 0.0f;
        float learning_rate = 0.1f;

        for (int epoch = 0; epoch < NUM_EPOCHS; ++epoch) {
            // Forward pass: Input -> Hidden
            hipdnnConvolutionForward(cudnn,
                                    &alpha,
                                    input_desc,
                                    d_X,
                                    filter_desc1,
                                    d_W1,
                                    conv_desc,
                                    HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                    d_workspace,
                                    workspace_bytes,
                                    &beta,
                                    hidden_desc,
                                    d_hidden);

            // Add bias
            hipdnnAddTensor(cudnn,
                           &alpha,
                           bias_desc1,
                           d_b1,
                           &alpha,
                           hidden_desc,
                           d_hidden);

            // Sigmoid activation
            hipdnnActivationForward(cudnn,
                                   activation_desc,
                                   &alpha,
                                   hidden_desc,
                                   d_hidden,
                                   &beta,
                                   hidden_desc,
                                   d_hidden);

            // Forward pass: Hidden -> Output
            hipdnnConvolutionForward(cudnn,
                                    &alpha,
                                    hidden_desc,
                                    d_hidden,
                                    filter_desc2,
                                    d_W2,
                                    conv_desc,
                                    HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                    d_workspace,
                                    workspace_bytes,
                                    &beta,
                                    output_desc,
                                    d_output);

            // Add bias
            hipdnnAddTensor(cudnn,
                           &alpha,
                           bias_desc2,
                           d_b2,
                           &alpha,
                           output_desc,
                           d_output);

            // Sigmoid activation
            hipdnnActivationForward(cudnn,
                                   activation_desc,
                                   &alpha,
                                   output_desc,
                                   d_output,
                                   &beta,
                                   output_desc,
                                   d_output);

            // Compute delta_output = d_output - d_Y
            hipdnnOpTensor(cudnn,
                          op_desc,
                          &alpha,
                          output_desc,
                          d_output,
                          &alpha,
                          output_desc, // Reuse output_desc instead of label_desc
                          d_Y,
                          &beta,
                          output_desc,
                          d_delta_output);

            // Backward pass through output activation
            hipdnnActivationBackward(cudnn,
                                    activation_desc,
                                    &alpha,
                                    output_desc,
                                    d_output,
                                    output_desc,
                                    d_delta_output,
                                    output_desc,
                                    d_output, // x (not used for sigmoid)
                                    &beta,
                                    output_desc,
                                    d_delta_output);

            // Compute gradients w.r.t W2 and b2
            hipdnnConvolutionBackwardFilter(cudnn,
                                           &alpha,
                                           hidden_desc,
                                           d_hidden,
                                           output_desc,
                                           d_delta_output,
                                           conv_desc,
                                           HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                           d_workspace,
                                           workspace_bytes,
                                           &beta,
                                           filter_desc2,
                                           d_dW2);

            hipdnnConvolutionBackwardBias(cudnn,
                                         &alpha,
                                         output_desc,
                                         d_delta_output,
                                         &beta,
                                         bias_desc2,
                                         d_db2);

            // Compute delta_hidden
            hipdnnConvolutionBackwardData(cudnn,
                                         &alpha,
                                         filter_desc2,
                                         d_W2,
                                         output_desc,
                                         d_delta_output,
                                         conv_desc,
                                         HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                         d_workspace,
                                         workspace_bytes,
                                         &beta,
                                         hidden_desc,
                                         d_delta_hidden);

            // Backward pass through hidden activation
            hipdnnActivationBackward(cudnn,
                                    activation_desc,
                                    &alpha,
                                    hidden_desc,
                                    d_hidden,
                                    hidden_desc,
                                    d_delta_hidden,
                                    hidden_desc,
                                    d_hidden, // x (not used for sigmoid)
                                    &beta,
                                    hidden_desc,
                                    d_delta_hidden);

            // Compute gradients w.r.t W1 and b1
            hipdnnConvolutionBackwardFilter(cudnn,
                                           &alpha,
                                           input_desc,
                                           d_X,
                                           hidden_desc,
                                           d_delta_hidden,
                                           conv_desc,
                                           HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                                           d_workspace,
                                           workspace_bytes,
                                           &beta,
                                           filter_desc1,
                                           d_dW1);

            hipdnnConvolutionBackwardBias(cudnn,
                                         &alpha,
                                         hidden_desc,
                                         d_delta_hidden,
                                         &beta,
                                         bias_desc1,
                                         d_db1);

            // Update weights and biases
            int threads_per_block = 256;
            int blocks_per_grid;

            blocks_per_grid = (input_size * hidden_size + threads_per_block - 1) / threads_per_block;
            update_weights<<<blocks_per_grid, threads_per_block>>>(d_W1, d_dW1, learning_rate, input_size * hidden_size);

            blocks_per_grid = (hidden_size * output_size + threads_per_block - 1) / threads_per_block;
            update_weights<<<blocks_per_grid, threads_per_block>>>(d_W2, d_dW2, learning_rate, hidden_size * output_size);

            blocks_per_grid = (hidden_size + threads_per_block - 1) / threads_per_block;
            update_weights<<<blocks_per_grid, threads_per_block>>>(d_b1, d_db1, learning_rate, hidden_size);

            blocks_per_grid = (output_size + threads_per_block - 1) / threads_per_block;
            update_weights<<<blocks_per_grid, threads_per_block>>>(d_b2, d_db2, learning_rate, output_size);
        }

        // End timing
        clock_t end = clock();
        double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
        printf("Time taken for N = %d: %f seconds\n", N, time_spent);

        // Cleanup
        hipFree(d_X);
        hipFree(d_Y);
        hipFree(d_W1);
        hipFree(d_b1);
        hipFree(d_W2);
        hipFree(d_b2);
        hipFree(d_hidden);
        hipFree(d_output);
        hipFree(d_delta_output);
        hipFree(d_delta_hidden);
        hipFree(d_dW1);
        hipFree(d_db1);
        hipFree(d_dW2);
        hipFree(d_db2);
        if (d_workspace)
            hipFree(d_workspace);

        free(h_X);
        free(h_Y);
        free(h_W1);
        free(h_b1);
        free(h_W2);
        free(h_b2);

        hipdnnDestroyOpTensorDescriptor(op_desc);
        hipdnnDestroyTensorDescriptor(bias_desc1);
        hipdnnDestroyTensorDescriptor(bias_desc2);
        hipdnnDestroyActivationDescriptor(activation_desc);
        hipdnnDestroyTensorDescriptor(input_desc);
        hipdnnDestroyTensorDescriptor(hidden_desc);
        hipdnnDestroyTensorDescriptor(output_desc);
        hipdnnDestroyFilterDescriptor(filter_desc1);
        hipdnnDestroyFilterDescriptor(filter_desc2);
        hipdnnDestroyConvolutionDescriptor(conv_desc);
        hipdnnDestroy(cudnn);
    }

    return 0;
}
